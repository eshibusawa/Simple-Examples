#include "hip/hip_runtime.h"

// This file is a modified version of CUB <https://github.com/NVIDIA/cub> example, see BSD license below.
//  BSD 2-Clause License
//
//  Copyright (c) 2023, Eijiro SHIBUSAWA
//  All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this
//     list of conditions and the following disclaimer.
//
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
//  DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
//  FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
//  DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
//  SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
//  OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
//  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>

//---------------------------------------------------------------------
// Kernels
//---------------------------------------------------------------------

/**
 * Simple kernel for performing a block-wide sorting over integers
 */
extern "C"
__launch_bounds__ (BLOCK_THREADS)
__global__ void BlockSortKernel(
	KEY_TYPE         *d_in,          // Tile of input
	KEY_TYPE         *d_out)         // Tile of output
{
	enum { TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD };

	// Specialize BlockLoad type for our thread block (uses warp-striped loads for coalescing, then transposes in shared memory to a blocked arrangement)
	typedef hipcub::BlockLoad<KEY_TYPE, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;

	// Specialize BlockRadixSort type for our thread block
	typedef hipcub::BlockRadixSort<KEY_TYPE, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

	// Shared memory
	__shared__ union TempStorage
	{
		typename BlockLoadT::TempStorage        load;
		typename BlockRadixSortT::TempStorage   sort;
	} temp_storage;

	// Per-thread tile items
	KEY_TYPE items[ITEMS_PER_THREAD];

	// Our current block's offset
	int block_offset = blockIdx.x * TILE_SIZE;

	// Load items into a blocked arrangement
	BlockLoadT(temp_storage.load).Load(d_in + block_offset, items);

	// Barrier for smem reuse
	__syncthreads();

	// Sort keys
	BlockRadixSortT(temp_storage.sort).SortBlockedToStriped(items);

	// Store output in striped fashion
	hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_out + block_offset, items);
}
