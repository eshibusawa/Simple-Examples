#include "hip/hip_runtime.h"
// BSD 2-Clause License
//
// Copyright (c) 2021, Eijiro SHIBUSAWA
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <torch/torch.h>

#include <cmath>

__constant__ float g_scale[1];

namespace {
inline int iDivUp(int a, int b)
{
	return static_cast<int>(std::ceil(static_cast<float>(a)/b));
}

template <typename scalar_t>
__global__ void scaleGridCUDAKernel(
	scalar_t* __restrict__ output,
	int height,
	int width)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	const int row = blockIdx.y * blockDim.y + threadIdx.y;

	if ((column >= width) || (row >= height))
	{
		return;
	}
	const int w2 = 2 * width, c2 = 2 * column;
	const int index = c2 + row * w2;
	output[index] = column * (g_scale[0]);
	output[index + 1] = row * (g_scale[0]);
}
}

torch::Tensor scaleGridCUDA(const std::vector<int> &outputSize, float scale)
{
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(g_scale), &scale, sizeof(float));
    if (hipSuccess != err)
	{
		std::cerr << "CUDA Error: " << __FILE__ << ":" << __LINE__ << std::endl;
		std::cerr << "   " << hipGetErrorString(err) << std::endl;
    }

	const int w = outputSize[1], h = outputSize[0];
	auto OutputOptions =
	torch::TensorOptions()
		.dtype(torch::kFloat)
		.device(torch::kCUDA, 0);
	auto output = torch::empty({h, w, 2}, OutputOptions);

	const dim3 threads(32, 32);
	const dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));
	AT_DISPATCH_FLOATING_TYPES(output.type(), "scaleGridCUDAKernelDispatch", ([&] {
		scaleGridCUDAKernel<scalar_t><<<blocks, threads>>>(
				output.data_ptr<scalar_t>(),
				h,
				w);
	}));

	return output;
}
