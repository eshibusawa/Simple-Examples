#include "hip/hip_runtime.h"
//  BSD 2-Clause License
//
//  Copyright (c) 2022, Eijiro SHIBUSAWA
//  All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this
//     list of conditions and the following disclaimer.
//
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
//  DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
//  FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
//  DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
//  SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
//  OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
//  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <mma.h>

extern "C" __global__ void wmma_16x16(const MatrixABType* __restrict__ a_ptr, const MatrixABType* __restrict__ b_ptr, MatrixCType* c_ptr)
{
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, MatrixABType, nvcuda::wmma::row_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, MatrixABType, nvcuda::wmma::row_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, MatrixCType> c_frag;

    nvcuda::wmma::load_matrix_sync(a_frag, a_ptr, 16);
    nvcuda::wmma::load_matrix_sync(b_frag, b_ptr, 16);
    nvcuda::wmma::load_matrix_sync(c_frag, c_ptr, 16, nvcuda::wmma::mem_row_major);

    // C = A*B + C
    nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

    nvcuda::wmma::store_matrix_sync(c_ptr, c_frag, 16, nvcuda::wmma::mem_row_major);
}
