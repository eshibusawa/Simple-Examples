#include "hip/hip_runtime.h"

//  BSD 2-Clause License
//
//  Copyright (c) 2025, Eijiro SHIBUSAWA
//  All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this
//     list of conditions and the following disclaimer.
//
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
//  DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
//  FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
//  DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
//  SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
//  OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
//  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#pragma pack(push, 4)
struct SimplePOD
{
	int x, y;
	float value;
};
#pragma pack(pop)

__constant__ SimplePOD g_POD[(CUSTOM_POD_NUMBER)] = {};

extern "C" __global__ void getPODSize(
	int *sz)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index > 0)
	{
		return;
	}
	*sz = sizeof(SimplePOD);
}

extern "C" __global__ void getPOD(
	SimplePOD *output)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= (CUSTOM_POD_NUMBER))
	{
		return;
	}
	output[index].x = g_POD[index].x;
	output[index].y = g_POD[index].y;
	output[index].value = g_POD[index].value;
}
